#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <hip/hip_runtime.h>

#include "cudaHelper.h"
#include "formula.h"

int main()
{
	// European calls to price, multiples of block size
	const int num_options = 1024;
	int num_dissteps;
	int num_blocks;
	int num_threads;

	// Record time via CUDA API
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Set test parameters to match reference
	double S0 = 7.7;
	double sigma = 0.3;
	double r = 0.04;
	double T = 16.0 / 365;
	double K = 7.51;
	double Smax = 50.0f;

	constexpr unsigned TIMESTEPS = 200;

    // Use Formula (not parallel)
    {
        double* resultGPU;
        checkCudaErrors(hipMalloc((void**)&resultGPU, sizeof(double)));
        hipEventRecord(start);
        UseFormula<<<1,1>>>(resultGPU, T, S0, r, K, sigma);
        double result;
        checkCudaErrors(hipMemcpy(&result, resultGPU, sizeof(double), hipMemcpyDeviceToHost));

        hipEventRecord(stop);
		hipEventSynchronize(stop);
		float millisecondsElapsed;
		hipEventElapsedTime(&millisecondsElapsed, start, stop);

        printf("Using Black-Scholes formula, result is: %lf\n", result);
        printf("Consumed %f ms\n", millisecondsElapsed);
    }

    // Perform explicit time-marching
    {
        // Parallel only by requiring multiple calculations
        printf("Explicit, single-precision\n");
    }

    // Perform implicit
	{
		printf("Implicit, single-precision, discretise S in to 514 slices, timestamps = 200\n");
        // Excludes end points Smin=0.0 and Smax=50.0
		num_dissteps = 512;
    }

    // Perform Crank Nicolson
	{
		printf("CrankNicolson, double-precision, discretise S in to 2050 slices, timesteps = 200\n");
        // Excludes end points Smin=0.0, Smax=50.0
        num_dissteps = 2048;
    }

    return 0;
}
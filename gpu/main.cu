#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <hip/hip_runtime.h>

#include "cudaHelper.h"
#include "formula.h"
#include "finiteDiff.h"

int main()
{
	// European calls to price, multiples of block size
	const int num_options = 1024;
	int num_dissteps;
	int num_blocks;
	int num_threads;

	// Record time via CUDA API
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Set test parameters to match reference
	double S0 = 7.7;
	double sigma = 0.3;
	double r = 0.04;
	double T = 16.0 / 365;
	double K = 7.51;
	double Smax = 50.0f;

	constexpr unsigned TIMESTEPS = 200;

    // Use Formula (not parallel)
    {
        double* resultGPU;
        checkCudaErrors(hipMalloc((void**)&resultGPU, sizeof(double)));
        hipEventRecord(start);
        UseFormula<<<1,1>>>(resultGPU, T, S0, r, K, sigma);
        double result;
        checkCudaErrors(hipMemcpy(&result, resultGPU, sizeof(double), hipMemcpyDeviceToHost));

        hipEventRecord(stop);
		hipEventSynchronize(stop);
		float millisecondsElapsed;
		hipEventElapsedTime(&millisecondsElapsed, start, stop);

        printf("Using Black-Scholes formula, result is: %lf\n", result);
        printf("Consumed %f ms\n", millisecondsElapsed);
    }

    // Perform explicit time-marching
    {
        printf("Explicit, single-precision\n");
        num_dissteps = 500;
        num_blocks = 1;
        num_threads = num_dissteps - 1;

        // Not parallel in time
        int memsize = (num_dissteps + 1) * sizeof(double);
		double* d_x;
		checkCudaErrors(hipMalloc((void**)&d_x, memsize));

        hipEventRecord(start);
        ExplicitMethod<<<num_blocks, num_threads>>>(sigma, Smax, K, T, r, d_x, TIMESTEPS, num_dissteps);
        double* x = (double*)malloc(memsize);
		checkCudaErrors(hipMemcpy(x, d_x, memsize, hipMemcpyDeviceToHost));

        hipEventRecord(stop);
		hipEventSynchronize(stop);
		float millisecondsElapsed;
		hipEventElapsedTime(&millisecondsElapsed, start, stop);

        // Get price by interpolation method
        double ds = Smax / (num_dissteps + 1);
        printf("Using Explicit method when spot price is %lf: %lf\n", S0, getPrice(S0, ds, x));
        printf("Consumed %f ms\n", millisecondsElapsed);

        hipFree(d_x);
        free(x);
    }

    // Perform implicit
	{
		printf("Implicit, single-precision, discretise S in to 514 slices, timestamps = 200\n");
        // Excludes end points Smin=0.0 and Smax=50.0
		num_dissteps = 512;

        // Compute the corresponding upper boundary. (we will treat Smin=0.0 in this case)
		float* upperbound;
		float* d_upperbound;
		upperbound = (float*)malloc((TIMESTEPS + 1) * sizeof(float));
		checkCudaErrors(hipMalloc((void**)&d_upperbound, (TIMESTEPS + 1) * sizeof(float)));
		for (int i = 0; i <= TIMESTEPS; i++)
		{
            // Vanilla European call option
			upperbound[i] = (float)((Smax - K) * exp(-r * T / TIMESTEPS * i));
		}
		checkCudaErrors(hipMemcpy(d_upperbound, upperbound, (TIMESTEPS + 1) * sizeof(float), hipMemcpyHostToDevice));

        // Allocate memory space for global array of solutions
		int memsize = num_options * num_dissteps * sizeof(float);
		float* d_x;
		checkCudaErrors(hipMalloc((void**)&d_x, memsize));

		hipEventRecord(start);
		num_threads = 256;
		num_blocks = num_options / 8;
        // perform the main time-marching of finite difference method, implicit scheme

		ImplicitMethod <<<num_blocks, num_threads>>>((float)sigma, (float)Smax, (float)K, (float)T, (float)r, d_x, d_upperbound);
		// copy results from device to host
		float* x = (float*)malloc(memsize);
		checkCudaErrors(hipMemcpy(x, d_x, memsize, hipMemcpyDeviceToHost));
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float millisecondsElapsed;
		hipEventElapsedTime(&millisecondsElapsed, start, stop);

		// Get the price by interpolation method
		float ds = (float)Smax / (num_dissteps + 1);
		printf("Price of test when spot price is %lf: %f\n", S0, getPrice((float)S0, ds, x));
		printf("Consumed %f ms\n", millisecondsElapsed / num_options);
		// cleanup memory
		checkCudaErrors(hipFree(d_x));
		checkCudaErrors(hipFree(d_upperbound));
		free(x);
		free(upperbound);
    }

    // Perform Crank Nicolson
	{
		printf("CrankNicolson, double-precision, discretise S in to 2050 slices, timesteps = 200\n");
        // Excludes end points Smin=0.0, Smax=50.0
		num_dissteps = 2048; //excluding end points Smin=0.0, Smax=50.0
        num_blocks = num_options;
		num_threads = num_dissteps / 2;


		// Compute the corresponding upper boundary. (we will treat Smin=0.0 in this case)
		double* upperbound, * d_upperbound;
		upperbound = (double*)malloc((TIMESTEPS + 1) * sizeof(double));
		checkCudaErrors(hipMalloc((void**)&d_upperbound, (TIMESTEPS + 1) * sizeof(double)));
		for (int i = 0; i <= TIMESTEPS; i++)
			upperbound[i] = (Smax - K) * exp(-r * T / TIMESTEPS * i);  //vanilla European call option	
		checkCudaErrors(hipMemcpy(d_upperbound, upperbound, (TIMESTEPS + 1) * sizeof(double), hipMemcpyHostToDevice));
        // allocate memory space for global arrays of coefficients
		int memsize = num_options * num_dissteps * sizeof(double);
		double* d_a1;
		double* d_b1;
		double* d_c1;
		double* d_x;
		checkCudaErrors(hipMalloc((void**)&d_a1, memsize / 2));
		checkCudaErrors(hipMalloc((void**)&d_b1, memsize / 2));
		checkCudaErrors(hipMalloc((void**)&d_c1, memsize / 2));
		checkCudaErrors(hipMalloc((void**)&d_x, memsize));

		hipEventRecord(start);
		CrankNicolsonMethod<<<num_blocks, num_threads, (num_dissteps + 1) * 5 * sizeof(float)>>>(sigma, Smax, K, T, r, d_a1, d_b1, d_c1, d_x, d_upperbound);

		// copy results from device to host
		double* x = (double*)malloc(memsize);
		checkCudaErrors(hipMemcpy(x, d_x, memsize, hipMemcpyDeviceToHost));

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float millisecondsElapsed;
		hipEventElapsedTime(&millisecondsElapsed, start, stop);

		// Get the price by interpolation method                        
		double ds = Smax / (num_dissteps + 1);

		printf("Price of test when spot price is %lf: %lf\n", S0, getPrice(S0, ds, x));
		printf("Consumed %f ms\n", millisecondsElapsed / num_options);

		// cleanup memory
		checkCudaErrors(hipFree(d_a1));
		checkCudaErrors(hipFree(d_b1));
		checkCudaErrors(hipFree(d_c1));
		checkCudaErrors(hipFree(d_x));
		checkCudaErrors(hipFree(d_upperbound));
		free(x);
		free(upperbound);
    }

    return 0;
}